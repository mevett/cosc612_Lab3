#include "hip/hip_runtime.h"

#include <wb.h>
#include "support.h"

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  Timer timer;
  hipError_t cuda_ret;

  args = wbArg_read(argc, argv);

  // Initialize host variables ----------------------------------------------

  printf("\nImporting data and creating memory on host..."); fflush(stdout);
  startTime(&timer);
  //wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
                            
  //@@ Set numCRows and numCColumns (to something other than 0, obviously)
  numCRows = 0;
  numCColumns = 0;
  //@@ Allocate the hostC matrix
  
  stopTime(&timer); printf("%f s\n", elapsedTime(timer));
  //wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  printf("Allocating GPU memory..."); fflush(stdout);
  startTime(&timer);
  //wbTime_start(GPU, "Allocating GPU memory.");
  
  //@@ Allocate GPU memory here

  stopTime(&timer); printf("%f s\n", elapsedTime(timer));
  //wbTime_stop(GPU, "Allocating GPU memory.");

  printf("Copying input memory to the GPU..."); fflush(stdout);
  startTime(&timer);
  //wbTime_start(GPU, "Copying input memory to the GPU.");
  
  //@@ Copy memory to the GPU here

  stopTime(&timer); printf("%f s\n", elapsedTime(timer));
  //wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here

  printf("Performing CUDA computation..."); fflush(stdout);
  startTime(&timer);
  //wbTime_start(Compute, "Performing CUDA computation");
  
  //@@ Launch the GPU Kernel here

  hipDeviceSynchronize();
  stopTime(&timer); printf("%f s\n", elapsedTime(timer));
  //wbTime_stop(Compute, "Performing CUDA computation");

  printf("Copying output memory to the CPU..."); fflush(stdout);
  startTime(&timer);
  //wbTime_start(Copy, "Copying output memory to the CPU");
  
  //@@ Copy the GPU memory back to the CPU here

  stopTime(&timer); printf("%f s\n", elapsedTime(timer));
  //wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("Freeing GPU Memory.."); fflush(stdout);
  startTime(&timer);
  //wbTime_start(GPU, "Freeing GPU Memory");
  
  //@@ Free the GPU memory here

  stopTime(&timer); printf("%f s\n", elapsedTime(timer));
  //wbTime_stop(GPU, "Freeing GPU Memory");

  //Determine if output is correct and print result.
  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
